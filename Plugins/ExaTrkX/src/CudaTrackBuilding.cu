// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include "Acts/Plugins/ExaTrkX/CudaTrackBuilding.hpp"
#include "Acts/Plugins/ExaTrkX/detail/ConnectedComponents.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.cuh"
#include "Acts/Utilities/Zip.hpp"

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/torch.h>

namespace Acts {

std::vector<std::vector<int>> CudaTrackBuilding::operator()(
    std::any /*nodes*/, std::any edges, std::any weights,
    std::vector<int>& spacepointIDs, const ExecutionContext& execContext) {
  ACTS_VERBOSE("Start CUDA track building");
  c10::cuda::CUDAStreamGuard guard(execContext.stream.value());

  const auto edgeTensor = std::any_cast<torch::Tensor>(edges).to(torch::kCUDA);
  assert(edgeTensor.size(0) == 2);

  const auto numSpacepoints = spacepointIDs.size();
  const auto numEdges = static_cast<std::size_t>(edgeTensor.size(1));

  if (numEdges == 0) {
    ACTS_WARNING("No edges remained after edge classification");
    return {};
  }

  auto stream = execContext.stream->stream();

  auto cudaSrcPtr = edgeTensor.data_ptr<std::int64_t>();
  auto cudaTgtPtr = edgeTensor.data_ptr<std::int64_t>() + numEdges;

  int* cudaLabels;
  ACTS_CUDA_CHECK(
      hipMallocAsync(&cudaLabels, numSpacepoints * sizeof(int), stream));

  std::size_t numberLabels = detail::connectedComponentsCuda(
      numEdges, cudaSrcPtr, cudaTgtPtr, numSpacepoints, cudaLabels, stream);

  // TODO not sure why there is an issue that is not detected in the unit tests
  numberLabels += 1;

  std::vector<int> trackLabels(numSpacepoints);
  ACTS_CUDA_CHECK(hipMemcpyAsync(trackLabels.data(), cudaLabels,
                                  numSpacepoints * sizeof(int),
                                  hipMemcpyDeviceToHost, stream));
  ACTS_CUDA_CHECK(hipFreeAsync(cudaLabels, stream));
  ACTS_CUDA_CHECK(hipStreamSynchronize(stream));
  ACTS_CUDA_CHECK(hipGetLastError());

  ACTS_VERBOSE("Found " << numberLabels << " track candidates");

  std::vector<std::vector<int>> trackCandidates(numberLabels);

  for (const auto [label, id] : Acts::zip(trackLabels, spacepointIDs)) {
    trackCandidates[label].push_back(id);
  }

  return trackCandidates;
}

}  // namespace Acts
