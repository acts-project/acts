// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include "Acts/Plugins/ExaTrkX/CudaTrackBuilding.hpp"
#include "Acts/Plugins/ExaTrkX/detail/ConnectedComponents.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.hpp"
#include "Acts/Plugins/ExaTrkX/detail/JunctionRemoval.hpp"
#include "Acts/Utilities/Zip.hpp"

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/torch.h>

namespace Acts {

std::vector<std::vector<int>> CudaTrackBuilding::operator()(
    std::any /*nodes*/, std::any edges, std::any weights,
    std::vector<int>& spacepointIDs, const ExecutionContext& execContext) {
  ACTS_VERBOSE("Start CUDA track building");

  const auto edgeTensor = std::any_cast<torch::Tensor>(edges).to(torch::kCUDA);
  assert(edgeTensor.size(0) == 2);

  const auto numSpacepoints = spacepointIDs.size();
  auto numEdges = static_cast<std::size_t>(edgeTensor.size(1));

  if (numEdges == 0) {
    ACTS_DEBUG("No edges remained after edge classification");
    return {};
  }

  auto stream = execContext.stream.value();

  auto cudaSrcPtr = edgeTensor.data_ptr<std::int64_t>();
  auto cudaTgtPtr = edgeTensor.data_ptr<std::int64_t>() + numEdges;

  auto ms = [](auto t0, auto t1) {
    return std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0)
        .count();
  };

  if (m_cfg.doJunctionRemoval) {
    const auto scoreTensor =
        std::any_cast<torch::Tensor>(weights).to(torch::kCUDA);
    assert(scoreTensor.size(0) == edgeTensor.size(1));
    auto cudaScorePtr = scoreTensor.data_ptr<float>();

    ACTS_DEBUG("Do junction removal...");
    auto t0 = std::chrono::high_resolution_clock::now();
    auto [cudaSrcPtrJr, numEdgesOut] = detail::junctionRemovalCuda(
        numEdges, numSpacepoints, cudaScorePtr, cudaSrcPtr, cudaTgtPtr, stream);
    auto t1 = std::chrono::high_resolution_clock::now();
    cudaSrcPtr = cudaSrcPtrJr;
    cudaTgtPtr = cudaSrcPtrJr + numEdgesOut;

    if (numEdgesOut == 0) {
      ACTS_WARNING(
          "No edges remained after junction removal, this should not happen!");
      ACTS_CUDA_CHECK(hipFreeAsync(cudaSrcPtrJr, stream));
      ACTS_CUDA_CHECK(hipStreamSynchronize(stream));
      return {};
    }

    ACTS_DEBUG("Removed " << numEdges - numEdgesOut
                          << " edges in junction removal");
    ACTS_DEBUG("Junction removal took " << ms(t0, t1) << " ms");
    numEdges = numEdgesOut;
  }

  int* cudaLabels{};
  ACTS_CUDA_CHECK(
      hipMallocAsync(&cudaLabels, numSpacepoints * sizeof(int), stream));

  auto t0 = std::chrono::high_resolution_clock::now();
  std::size_t numberLabels = detail::connectedComponentsCuda(
      numEdges, cudaSrcPtr, cudaTgtPtr, numSpacepoints, cudaLabels, stream,
      m_cfg.useOneBlockImplementation);
  auto t1 = std::chrono::high_resolution_clock::now();
  ACTS_DEBUG("Connected components took " << ms(t0, t1) << " ms");

  // TODO not sure why there is an issue that is not detected in the unit tests
  numberLabels += 1;

  std::vector<int> trackLabels(numSpacepoints);
  ACTS_CUDA_CHECK(hipMemcpyAsync(trackLabels.data(), cudaLabels,
                                  numSpacepoints * sizeof(int),
                                  hipMemcpyDeviceToHost, stream));

  // Free Memory
  ACTS_CUDA_CHECK(hipFreeAsync(cudaLabels, stream));
  if (m_cfg.doJunctionRemoval) {
    ACTS_CUDA_CHECK(hipFreeAsync(cudaSrcPtr, stream));
  }

  ACTS_CUDA_CHECK(hipStreamSynchronize(stream));
  ACTS_CUDA_CHECK(hipGetLastError());

  ACTS_VERBOSE("Found " << numberLabels << " track candidates");

  std::vector<std::vector<int>> trackCandidates(numberLabels);

  for (const auto [label, id] : Acts::zip(trackLabels, spacepointIDs)) {
    trackCandidates[label].push_back(id);
  }

  return trackCandidates;
}

}  // namespace Acts
