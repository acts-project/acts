#include "hip/hip_runtime.h"
// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include "Acts/Plugins/ExaTrkX/Tensor.hpp"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.hpp"

#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

namespace {

__global__ void sigmoidImpl(std::size_t size, float *array) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) {
    return;
  }

  array[i] = 1.f / (1.f + __expf(-array[i]));
}

__global__ void applyCut(std::size_t size, float cutoff, const float *array,
                         bool *mask) {
  std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) {
    return;
  }

  mask[i] = array[i] > cutoff;
}

}  // namespace

namespace Acts::detail {

void cudaSigmoid(Tensor<float> &tensor, hipStream_t stream) {
  dim3 blockDim = 1024;
  dim3 gridDim = (tensor.size() + blockDim.x - 1) / blockDim.x;
  sigmoidImpl<<<gridDim, blockDim, 0, stream>>>(tensor.size(), tensor.data());
  ACTS_CUDA_CHECK(hipGetLastError());
}

std::pair<Tensor<float>, Tensor<std::int64_t>> cudaApplyScoreCut(
    const Tensor<float> &scores, const Tensor<std::int64_t> &edgeIndex,
    float cut, hipStream_t stream) {
  dim3 blockDim = 1024;
  dim3 gridDim = (scores.size() + blockDim.x - 1) / blockDim.x;
  ExecutionContext execContext{scores.device(), stream};

  bool *mask{};
  ACTS_CUDA_CHECK(hipMallocAsync(&mask, scores.size() * sizeof(bool), stream));

  applyCut<<<gridDim, blockDim, 0, stream>>>(scores.size(), cut, scores.data(),
                                             mask);
  ACTS_CUDA_CHECK(hipGetLastError());

  const std::size_t nEdgesAfter = thrust::count(thrust::device.on(stream), mask,
                                                mask + scores.size(), true);

  auto outputScores = Tensor<float>::Create({nEdgesAfter, 1}, execContext);
  auto outputEdgeIndex =
      Tensor<std::int64_t>::Create({2, nEdgesAfter}, execContext);

  auto pred = [] __device__(bool x) { return x; };
  thrust::copy_if(thrust::device.on(stream), scores.data(),
                  scores.data() + scores.size(), mask, outputScores.data(),
                  pred);

  const auto edgesBefore = edgeIndex.size() / 2;
  thrust::copy_if(thrust::device.on(stream), edgeIndex.data(),
                  edgeIndex.data() + edgesBefore, mask, outputEdgeIndex.data(),
                  pred);
  thrust::copy_if(thrust::device.on(stream), edgeIndex.data() + edgesBefore,
                  edgeIndex.data() + 2 * edgesBefore, mask,
                  outputEdgeIndex.data() + nEdgesAfter, pred);

  ACTS_CUDA_CHECK(hipFreeAsync(mask, *execContext.stream));
  return {std::move(outputScores), std::move(outputEdgeIndex)};
}

}  // namespace Acts::detail
