#include "hip/hip_runtime.h"
// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.cuh"
#include "Acts/Plugins/ExaTrkX/detail/CudaUtils.hpp"
#include "Acts/Plugins/ExaTrkX/detail/JunctionRemoval.hpp"

#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <thrust/transform_scan.h>

namespace Acts::detail {

__global__ void findNumInOutEdge(std::size_t nEdges,
                                 const std::int64_t *srcNodes,
                                 const std::int64_t *dstNodes, int *numInEdges,
                                 int *numOutEdges) {
  const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nEdges) {
    return;
  }

  auto srcNode = srcNodes[i];
  auto dstNode = dstNodes[i];

  atomicAdd(&numInEdges[dstNode], 1);
  atomicAdd(&numOutEdges[srcNode], 1);
}

__global__ void fillJunctionEdges(std::size_t nEdges,
                                  const std::int64_t *edgeNodes,
                                  const int *numEdgesPrefixSum,
                                  int *junctionEdges, int *junctionEdgeOffset) {
  const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nEdges) {
    return;
  }

  int node = edgeNodes[i];
  int base = numEdgesPrefixSum[node];
  int numEdgesNode = numEdgesPrefixSum[node + 1] - base;

  // Zero is allowed, because we set 1 to 0 before
  assert(numEdgesNode != 1 && "node is not a junction");

  if (numEdgesNode != 0) {
    int offset = atomicAdd(&junctionEdgeOffset[node], 1);
    assert(offset < numEdgesNode && "inconsistent offset with number of edges");
    junctionEdges[base + offset] = i;
  }
}

__global__ void fillEdgeMask(std::size_t nNodes, const float *scores,
                             const int *numEdgesPrefixSum,
                             const int *junctionEdges,
                             bool *edgesToRemoveMask) {
  const std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nNodes) {
    return;
  }

  // Get the bse and number of edges for the current node
  int base = numEdgesPrefixSum[i];
  int numEdgesNode = numEdgesPrefixSum[i + 1] - base;

  // Find the edge with the maximum score
  float maxScore = 0.0f;
  int edgeIdMaxScore = -1;
  for (int j = base; j < base + numEdgesNode; ++j) {
    int edgeId = junctionEdges[j];
    float score = scores[edgeId];
    if (score > maxScore) {
      maxScore = score;
      edgeIdMaxScore = edgeId;
    }
  }

  // Mark all edges except the one with the maximum score for removal
  for (int j = base; j < base + numEdgesNode; ++j) {
    int edgeId = junctionEdges[j];
    if (edgeId != edgeIdMaxScore) {
      edgesToRemoveMask[edgeId] = true;
    }
  }
}

struct LogicalNotPredicate {
  bool __device__ operator()(bool b) { return !b; }
};

// When we perform the prefix sum over the number of outgoing/incoming edges,
// we only want to count edges that are part of a junction. The requirement for
// this is that there are >= 2 outgoing/incoming edges.
// Therefore, we design the accumulation operator in a way, that it returns zero
// for non-junction cases
// This allows to skip a preprocessing step to set the edge count for
// non-junction nodes explicitly to zero Also, it should work for the prefix sum
// values, since those will only ever be 0 or >= 2 due to the above requirements
// A informal proof of associativity is given in
// https://github.com/acts-project/acts/pull/4223
struct AccumulateJunctionEdges {
  int __device__ operator()(int a, int b) const {
    a = a < 2 ? 0 : a;
    b = b < 2 ? 0 : b;
    return a + b;
  }
};

std::pair<std::int64_t *, std::size_t> junctionRemovalCuda(
    std::size_t nEdges, std::size_t nNodes, const float *scores,
    const std::int64_t *srcNodes, const std::int64_t *dstNodes,
    hipStream_t stream) {
  // Allocate device memory for the number of in and out edges
  int *numInEdges{}, *numOutEdges{};
  ACTS_CUDA_CHECK(
      hipMallocAsync(&numInEdges, (nNodes + 1) * sizeof(int), stream));
  ACTS_CUDA_CHECK(
      hipMallocAsync(&numOutEdges, (nNodes + 1) * sizeof(int), stream));

  // Initialize the number of in and out edges to 0
  ACTS_CUDA_CHECK(
      hipMemsetAsync(numInEdges, 0, (nNodes + 1) * sizeof(int), stream));
  ACTS_CUDA_CHECK(
      hipMemsetAsync(numOutEdges, 0, (nNodes + 1) * sizeof(int), stream));

  // Launch the kernel to find the number of in and out edges
  const dim3 blockSize = 512;
  const dim3 gridSizeEdges = (nEdges + blockSize.x - 1) / blockSize.x;
  findNumInOutEdge<<<gridSizeEdges, blockSize, 0, stream>>>(
      nEdges, srcNodes, dstNodes, numInEdges, numOutEdges);
  ACTS_CUDA_CHECK(hipGetLastError());

  // Perform prefix sum on the number of in and out edges with a special
  // reduction that does not include edges from non-junction nodes
  thrust::exclusive_scan(thrust::device.on(stream), numInEdges,
                         numInEdges + nNodes + 1, numInEdges, 0,
                         AccumulateJunctionEdges{});
  thrust::exclusive_scan(thrust::device.on(stream), numOutEdges,
                         numOutEdges + nNodes + 1, numOutEdges, 0,
                         AccumulateJunctionEdges{});

  // Find the total number of in and out edges involved in junctions
  int numJunctionInEdges{}, numJunctionOutEdges{};
  ACTS_CUDA_CHECK(hipMemcpyAsync(&numJunctionInEdges, &numInEdges[nNodes],
                                  sizeof(int), hipMemcpyDeviceToHost, stream));
  ACTS_CUDA_CHECK(hipMemcpyAsync(&numJunctionOutEdges, &numOutEdges[nNodes],
                                  sizeof(int), hipMemcpyDeviceToHost, stream));
  ACTS_CUDA_CHECK(hipStreamSynchronize(stream));

  // Allocate device memory to store the edge indices for in and out edges
  int *junctionInEdges{}, *junctionOutEdges{};
  ACTS_CUDA_CHECK(hipMallocAsync(&junctionInEdges,
                                  numJunctionInEdges * sizeof(int), stream));
  ACTS_CUDA_CHECK(hipMallocAsync(&junctionOutEdges,
                                  numJunctionOutEdges * sizeof(int), stream));

  // Allocate device memory for the running index of the in and out edges per
  // node
  int *junctionInEdgeOffset{}, *junctionOutEdgeOffset{};
  ACTS_CUDA_CHECK(
      hipMallocAsync(&junctionInEdgeOffset, nNodes * sizeof(int), stream));
  ACTS_CUDA_CHECK(
      hipMallocAsync(&junctionOutEdgeOffset, nNodes * sizeof(int), stream));
  ACTS_CUDA_CHECK(
      hipMemsetAsync(junctionInEdgeOffset, 0, nNodes * sizeof(int), stream));
  ACTS_CUDA_CHECK(
      hipMemsetAsync(junctionOutEdgeOffset, 0, nNodes * sizeof(int), stream));

  // Fill the junction edges for in and out edges
  fillJunctionEdges<<<gridSizeEdges, blockSize, 0, stream>>>(
      nEdges, srcNodes, numOutEdges, junctionOutEdges, junctionOutEdgeOffset);
  ACTS_CUDA_CHECK(hipGetLastError());
  fillJunctionEdges<<<gridSizeEdges, blockSize, 0, stream>>>(
      nEdges, dstNodes, numInEdges, junctionInEdges, junctionInEdgeOffset);
  ACTS_CUDA_CHECK(hipGetLastError());

  // Allocate device memory for the edge mask
  bool *edgesToRemoveMask{};
  ACTS_CUDA_CHECK(
      hipMallocAsync(&edgesToRemoveMask, nEdges * sizeof(bool), stream));
  ACTS_CUDA_CHECK(
      hipMemsetAsync(edgesToRemoveMask, 0, nEdges * sizeof(bool), stream));

  // Fill the edge mask with the edges to be removed
  const dim3 gridSizeNodes = (nNodes + blockSize.x - 1) / blockSize.x;
  fillEdgeMask<<<gridSizeNodes, blockSize, 0, stream>>>(
      nNodes, scores, numInEdges, junctionInEdges, edgesToRemoveMask);
  ACTS_CUDA_CHECK(hipGetLastError());
  fillEdgeMask<<<gridSizeNodes, blockSize, 0, stream>>>(
      nNodes, scores, numOutEdges, junctionOutEdges, edgesToRemoveMask);
  ACTS_CUDA_CHECK(hipGetLastError());

  // Free the device memory
  ACTS_CUDA_CHECK(hipFreeAsync(numInEdges, stream));
  ACTS_CUDA_CHECK(hipFreeAsync(numOutEdges, stream));
  ACTS_CUDA_CHECK(hipFreeAsync(junctionInEdges, stream));
  ACTS_CUDA_CHECK(hipFreeAsync(junctionOutEdges, stream));
  ACTS_CUDA_CHECK(hipFreeAsync(junctionInEdgeOffset, stream));
  ACTS_CUDA_CHECK(hipFreeAsync(junctionOutEdgeOffset, stream));

  // Compactify the edges based on the edge mask
  int nEdgesToRemove =
      thrust::count(thrust::device.on(stream), edgesToRemoveMask,
                    edgesToRemoveMask + nEdges, true);
  int nEdgesAfter = nEdges - nEdgesToRemove;
  // Allocate memory for the new srcNodes and dstNodes arrays
  std::int64_t *newSrcNodes{};
  ACTS_CUDA_CHECK(hipMallocAsync(
      &newSrcNodes, 2 * nEdgesAfter * sizeof(std::int64_t), stream));
  std::int64_t *newDstNodes = newSrcNodes + nEdgesAfter;

  // Compactify the srcNodes and dstNodes arrays based on the edge mask
  thrust::copy_if(thrust::device.on(stream), srcNodes, srcNodes + nEdges,
                  edgesToRemoveMask, newSrcNodes, LogicalNotPredicate{});
  thrust::copy_if(thrust::device.on(stream), dstNodes, dstNodes + nEdges,
                  edgesToRemoveMask, newDstNodes, LogicalNotPredicate{});

  // Free the device memory for the edge mask
  ACTS_CUDA_CHECK(hipFreeAsync(edgesToRemoveMask, stream));

  // Synchronize the stream
  ACTS_CUDA_CHECK(hipStreamSynchronize(stream));

  return std::make_pair(newSrcNodes, static_cast<std::size_t>(nEdgesAfter));
}

}  // namespace Acts::detail
