#include "hip/hip_runtime.h"
// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

// CUDA plugin include(s).
#include "Acts/Plugins/Cuda/Seeding2/Details/FindTriplets.hpp"
#include "Acts/Plugins/Cuda/Seeding2/Details/Types.hpp"
#include "Acts/Plugins/Cuda/Seeding2/TripletFilterConfig.hpp"
#include "Acts/Plugins/Cuda/Utilities/MemoryManager.hpp"

#include "../Utilities/ErrorCheck.cuh"
#include "../Utilities/MatrixMacros.hpp"

// Acts include(s).
#include "Acts/Seeding/SeedFilterConfig.hpp"

// CUDA include(s).
#include <hip/hip_runtime.h>

// System include(s).
#include <cassert>
#include <cmath>
#include <cstring>

namespace Acts {
namespace Cuda {
namespace Kernels {

/// Function performing coordinate transformation for one spacepoint pair
///
/// @param spM    The middle spacepoint to use
/// @param sp     The "other" spacepoint to use
/// @param bottom @c true If the "other" spacepoint is a bottom one, @c false
///               otherwise
__device__ Details::LinCircle transformCoordinates(
    const Details::SpacePoint& spM, const Details::SpacePoint& sp,
    bool bottom) {
  // Create the result object.
  Details::LinCircle result;

  // Parameters of the middle spacepoint.
  const float cosPhiM = spM.x / spM.radius;
  const float sinPhiM = spM.y / spM.radius;

  // (Relative) Parameters of the spacepoint being transformed.
  const float deltaX = sp.x - spM.x;
  const float deltaY = sp.y - spM.y;
  const float deltaZ = sp.z - spM.z;

  // calculate projection fraction of spM->sp vector pointing in same
  // direction as
  // vector origin->spM (x) and projection fraction of spM->sp vector pointing
  // orthogonal to origin->spM (y)
  const float x = deltaX * cosPhiM + deltaY * sinPhiM;
  const float y = deltaY * cosPhiM - deltaX * sinPhiM;
  // 1/(length of M -> SP)
  const float iDeltaR2 = 1. / (deltaX * deltaX + deltaY * deltaY);
  const float iDeltaR = sqrtf(iDeltaR2);
  //
  const int bottomFactor = 1 * (int(!bottom)) - 1 * (int(bottom));
  // cot_theta = (deltaZ/deltaR)
  const float cot_theta = deltaZ * iDeltaR * bottomFactor;
  // VERY frequent (SP^3) access
  result.cotTheta = cot_theta;
  // location on z-axis of this SP-duplet
  result.Zo = spM.z - spM.radius * cot_theta;
  result.iDeltaR = iDeltaR;
  // transformation of circle equation (x,y) into linear equation (u,v)
  // x^2 + y^2 - 2x_0*x - 2y_0*y = 0
  // is transformed into
  // 1 - 2x_0*u - 2y_0*v = 0
  // using the following m_U and m_V
  // (u = A + B*v); A and B are created later on
  result.U = x * iDeltaR2;
  result.V = y * iDeltaR2;
  // error term for sp-pair without correlation of middle space point
  result.Er = ((spM.varianceZ + sp.varianceZ) +
               (cot_theta * cot_theta) * (spM.varianceR + sp.varianceR)) *
              iDeltaR2;
  return result;
}

/// Kernel performing coordinate transformation on all created dublets
///
/// @param[in] nDublets The total number of dublets found
/// @param[in] maxMBDublets The maximal number of middle-bottom dublets found
///            for any middle spacepoint
/// @param[in] maxMTDublets The maximal number of middle-top dublets found for
///            any middle spacepoint
/// @param[in] nBottomSPs The number of bottom spacepoints in @c bottomSPs
/// @param[in] bottomSPs Properties of all of the bottom spacepoints
/// @param[in] nMiddleSPs The number of middle spacepoints in @c middleSPs
/// @param[in] middleSPs Properties of all of the middle spacepoints
/// @param[in] nTopSPs The number of top spacepoints in @c topSPs
/// @param[in] topSPs Properties of all of the top spacepoints
/// @param[in] middleBottomCounts 1-D array of the number of middle-bottom
///            dublets found for each middle spacepoint
/// @param[in] middleBottomDublets 2-D matrix of size
///            @c nMiddleSPs x @c nBottomSPs, holding the bottom spacepoint
///            indices for the identified middle-bottom dublets
/// @param[in] middleTopCounts 1-D array of the number of middle-top dublets
///            found for each middle spacepoint
/// @param[in] middleTopDublets 2-D matrix of size
///            @c nMiddleSPs x @c nTopSPs, holding the top spacepoint
///            indices for the identified middle-top dublets
/// @param[out] bottomSPLinTransArray 2-dimensional matrix indexed the same way
///             as @c middleBottomDublets
/// @param[out] topSPLinTransArray 2-dimensional matrix indexed the same way as
///             @c middleTopDublets
///
__global__ void transformCoordinates(
    unsigned int nDublets, unsigned int maxMBDublets, unsigned int maxMTDublets,
    std::size_t nBottomSPs, const Details::SpacePoint* bottomSPs,
    std::size_t nMiddleSPs, const Details::SpacePoint* middleSPs,
    std::size_t nTopSPs, const Details::SpacePoint* topSPs,
    const unsigned int* middleBottomCounts,
    const std::size_t* middleBottomDublets, const unsigned int* middleTopCounts,
    const std::size_t* middleTopDublets,
    Details::LinCircle* bottomSPLinTransArray,
    Details::LinCircle* topSPLinTransArray) {
  // Get the global index.
  const int dubletIndex = blockIdx.x * blockDim.x + threadIdx.x;

  // If we're out of bounds, finish right away.
  if (dubletIndex >= nDublets) {
    return;
  }

  // Find the dublet to transform.
  std::size_t middleIndex = 0;
  int runningIndex = dubletIndex;
  int tmpValue = 0;
  while (runningIndex >= (tmpValue = (middleBottomCounts[middleIndex] +
                                      middleTopCounts[middleIndex]))) {
    middleIndex += 1;
    assert(middleIndex < nMiddleSPs);
    runningIndex -= tmpValue;
  }
  const bool transformBottom =
      ((runningIndex < middleBottomCounts[middleIndex]) ? true : false);
  const std::size_t bottomMatrixIndex = (transformBottom ? runningIndex : 0);
  const std::size_t topMatrixIndex =
      (transformBottom ? 0 : runningIndex - middleBottomCounts[middleIndex]);

  // Perform the transformation.
  if (transformBottom) {
    const std::size_t bottomIndex =
        ACTS_CUDA_MATRIX2D_ELEMENT(middleBottomDublets, nMiddleSPs, nBottomSPs,
                                   middleIndex, bottomMatrixIndex);
    assert(bottomIndex < nBottomSPs);
    ACTS_CUDA_MATRIX2D_ELEMENT(bottomSPLinTransArray, nMiddleSPs, maxMBDublets,
                               middleIndex, bottomMatrixIndex) =
        transformCoordinates(middleSPs[middleIndex], bottomSPs[bottomIndex],
                             true);
  } else {
    const std::size_t topIndex = ACTS_CUDA_MATRIX2D_ELEMENT(
        middleTopDublets, nMiddleSPs, nTopSPs, middleIndex, topMatrixIndex);
    assert(topIndex < nTopSPs);
    ACTS_CUDA_MATRIX2D_ELEMENT(topSPLinTransArray, nMiddleSPs, maxMTDublets,
                               middleIndex, topMatrixIndex) =
        transformCoordinates(middleSPs[middleIndex], topSPs[topIndex], false);
  }

  return;
}

/// Kernel used for finding all the triplet candidates
///
/// @param[in] middleIndexStart The middle spacepoint index that the kernel was
///            "started from"
/// @param[in] maxMBDublets The maximal number of middle-bottom dublets found
///            for any middle spacepoint
/// @param[in] maxMTDublets The maximal number of middle-top dublets found for
///            any middle spacepoint
/// @param[in] maxTriplets The maximum number of triplets for which memory is
///            booked
/// @param[in] nParallelMiddleSPs The number of middle spacepoints that the
///            "largest" kernels may be started on in parallel
/// @param[in] nMiddleSPsProcessed The number of middle spacepoints that the
///            kernel was started on in parallel
/// @param[in] nBottomSPs The number of bottom spacepoints in @c bottomSPs
/// @param[in] bottomSPs Properties of all of the bottom spacepoints
/// @param[in] nMiddleSPs The number of middle spacepoints in @c middleSPs
/// @param[in] middleSPs Properties of all of the middle spacepoints
/// @param[in] nTopSPs The number of top spacepoints in @c topSPs
/// @param[in] topSPs Properties of all of the top spacepoints
/// @param[in] middleBottomCounts 1-D array of the number of middle-bottom
///            dublets found for each middle spacepoint
/// @param[in] middleBottomDublets 2-D matrix of size
///            @c nMiddleSPs x @c nBottomSPs, holding the bottom spacepoint
///            indices for the identified middle-bottom dublets
/// @param[in] middleTopCounts 1-D array of the number of middle-top dublets
///            found for each middle spacepoint
/// @param[in] middleTopDublets 2-D matrix of size
///            @c nMiddleSPs x @c nTopSPs, holding the top spacepoint
///            indices for the identified middle-top dublets
/// @param[in] bottomSPLinTransArray 2-dimensional matrix indexed the same way
///            as @c middleBottomArray
/// @param[in] topSPLinTransArray 2-dimensional matrix indexed the same way as
///            @c middleTopArray
/// @param[in] maxScatteringAngle2 Parameter from @c Acts::SeedFinderConfig
/// @param[in] sigmaScattering Parameter from @c Acts::SeedFinderConfig
/// @param[in] minHelixDiameter2 Parameter from @c Acts::SeedFinderConfig
/// @param[in] pT2perRadius Parameter from @c Acts::SeedFinderConfig
/// @param[in] impactMax Parameter from @c Acts::SeedFinderConfig
/// @param[in] impactWeightFactor Parameter from @c Acts::SeedFinderConfig
/// @param[out] tripletsPerBottomDublet 1-dimensional array of the triplet
///             counts for each bottom spacepoint
/// @param[out] tripletIndices 2-dimensional matrix of the indices of the
///             triplets created for each middle-bottom spacepoint dublet
/// @param[out] maxTripletsPerSpB Pointer to the scalar outputting the maximum
///             number of triplets found for any bottom spacepoint dublet
/// @param[out] tripletCount Pointer to the scalar counting the total number of
///             triplets created by the kernel
/// @param[out] triplets 1-dimensional array of all reconstructed triplet
///             candidates
///
__global__ void findTriplets(
    std::size_t middleIndexStart, unsigned int maxMBDublets,
    unsigned int maxMTDublets, unsigned int maxTriplets,
    std::size_t nParallelMiddleSPs, std::size_t nMiddleSPsProcessed,
    std::size_t nBottomSPs, const Details::SpacePoint* bottomSPs,
    std::size_t nMiddleSPs, const Details::SpacePoint* middleSPs,
    std::size_t nTopSPs, const Details::SpacePoint* topSPs,
    const unsigned int* middleBottomCounts,
    const std::size_t* middleBottomDublets, const unsigned int* middleTopCounts,
    const std::size_t* middleTopDublets,
    const Details::LinCircle* bottomSPLinTransArray,
    const Details::LinCircle* topSPLinTransArray, float maxScatteringAngle2,
    float sigmaScattering, float minHelixDiameter2, float pT2perRadius,
    float impactMax, float impactWeightFactor,
    unsigned int* tripletsPerBottomDublet, std::size_t* tripletIndices,
    unsigned int* maxTripletsPerSpB, unsigned int* tripletCount,
    Details::Triplet* triplets) {
  // A sanity check.
  assert(middleIndexStart + nMiddleSPsProcessed <= nMiddleSPs);

  // Find the middle spacepoint index to operate on.
  const unsigned int middleIndexOffset = blockIdx.x * blockDim.x + threadIdx.x;
  if (middleIndexOffset >= nMiddleSPsProcessed) {
    return;
  }
  const unsigned int middleIndex = middleIndexStart + middleIndexOffset;
  assert(middleIndex < nMiddleSPs);

  // Counts of middle-bottom and middle-top pairs for this middle spacepoint.
  const unsigned int middleBottomPairCount = middleBottomCounts[middleIndex];
  const unsigned int middleTopPairCount = middleTopCounts[middleIndex];

  // Find the indices of the middle-bottom and middle-top pairs to operate on.
  const unsigned int tripletCandidateIndex =
      blockIdx.y * blockDim.y + threadIdx.y;
  if (tripletCandidateIndex >= middleBottomPairCount * middleTopPairCount) {
    return;
  }
  const unsigned int bottomDubletIndex =
      tripletCandidateIndex / middleTopPairCount;
  assert(bottomDubletIndex < middleBottomPairCount);
  const unsigned int topDubletIndex =
      tripletCandidateIndex - bottomDubletIndex * middleTopPairCount;
  assert(topDubletIndex < middleTopPairCount);

  // Get the indices of the spacepoints to operate on.
  const unsigned int bottomIndex =
      ACTS_CUDA_MATRIX2D_ELEMENT(middleBottomDublets, nMiddleSPs, nBottomSPs,
                                 middleIndex, bottomDubletIndex);
  assert(bottomIndex < nBottomSPs);
  const unsigned int topIndex = ACTS_CUDA_MATRIX2D_ELEMENT(
      middleTopDublets, nMiddleSPs, nTopSPs, middleIndex, topDubletIndex);
  assert(topIndex < nTopSPs);

  // Load the transformed coordinates of the bottom spacepoint into the thread.
  const Details::LinCircle lb =
      ACTS_CUDA_MATRIX2D_ELEMENT(bottomSPLinTransArray, nMiddleSPs,
                                 maxMBDublets, middleIndex, bottomDubletIndex);

  // 1+(cot^2(theta)) = 1/sin^2(theta)
  float iSinTheta2 = (1. + lb.cotTheta * lb.cotTheta);
  // calculate max scattering for min momentum at the seed's theta angle
  // scaling scatteringAngle^2 by sin^2(theta) to convert pT^2 to p^2
  // accurate would be taking 1/atan(thetaBottom)-1/atan(thetaTop) <
  // scattering
  // but to avoid trig functions we approximate cot by scaling by
  // 1/sin^4(theta)
  // resolving with pT to p scaling --> only divide by sin^2(theta)
  // max approximation error for allowed scattering angles of 0.04 rad at
  // eta=infinity: ~8.5%
  float scatteringInRegion2 = maxScatteringAngle2 * iSinTheta2;
  // multiply the squared sigma onto the squared scattering
  scatteringInRegion2 *= sigmaScattering * sigmaScattering;

  // Load the transformed coordinates of the top spacepoint into the thread.
  const Details::LinCircle lt =
      ACTS_CUDA_MATRIX2D_ELEMENT(topSPLinTransArray, nMiddleSPs, maxMTDublets,
                                 middleIndex, topDubletIndex);

  // Load the parameters of the middle spacepoint into the thread.
  const Details::SpacePoint spM = middleSPs[middleIndex];

  // add errors of spB-spM and spM-spT pairs and add the correlation term
  // for errors on spM
  float error2 =
      lt.Er + lb.Er +
      2 * (lb.cotTheta * lt.cotTheta * spM.varianceR + spM.varianceZ) *
          lb.iDeltaR * lt.iDeltaR;

  float deltaCotTheta = lb.cotTheta - lt.cotTheta;
  float deltaCotTheta2 = deltaCotTheta * deltaCotTheta;
  float dCotThetaMinusError2 = 0.0f;

  // if the error is larger than the difference in theta, no need to
  // compare with scattering
  if (deltaCotTheta2 - error2 > 0) {
    deltaCotTheta = fabs(deltaCotTheta);
    // if deltaTheta larger than the scattering for the lower pT cut, skip
    float error = sqrtf(error2);
    dCotThetaMinusError2 = deltaCotTheta2 + error2 - 2 * deltaCotTheta * error;
    // avoid taking root of scatteringInRegion
    // if left side of ">" is positive, both sides of inequality can be
    // squared
    // (scattering is always positive)
    if (dCotThetaMinusError2 > scatteringInRegion2) {
      return;
    }
  }

  // protects against division by 0
  float dU = lt.U - lb.U;
  if (dU == 0.) {
    return;
  }
  // A and B are evaluated as a function of the circumference parameters
  // x_0 and y_0
  float A = (lt.V - lb.V) / dU;
  float S2 = 1. + A * A;
  float B = lb.V - A * lb.U;
  float B2 = B * B;
  // sqrt(S2)/B = 2 * helixradius
  // calculated radius must not be smaller than minimum radius
  if (S2 < B2 * minHelixDiameter2) {
    return;
  }
  // 1/helixradius: (B/sqrt(S2))/2 (we leave everything squared)
  float iHelixDiameter2 = B2 / S2;
  // calculate scattering for p(T) calculated from seed curvature
  float pT2scatter = 4 * iHelixDiameter2 * pT2perRadius;
  // TODO: include upper pT limit for scatter calc
  // convert p(T) to p scaling by sin^2(theta) AND scale by 1/sin^4(theta)
  // from rad to deltaCotTheta
  float p2scatter = pT2scatter * iSinTheta2;
  // if deltaTheta larger than allowed scattering for calculated pT, skip
  if ((deltaCotTheta2 - error2 > 0) &&
      (dCotThetaMinusError2 > p2scatter * sigmaScattering * sigmaScattering)) {
    return;
  }
  // A and B allow calculation of impact params in U/V plane with linear
  // function
  // (in contrast to having to solve a quadratic function in x/y plane)
  float Im = fabs((A - B * spM.radius) * spM.radius);

  // Check if the triplet candidate should be accepted.
  if (Im > impactMax) {
    return;
  }

  // Reserve elements (positions) in the global matrices/arrays.
  unsigned int* tripletIndexRowPtr = &(ACTS_CUDA_MATRIX2D_ELEMENT(
      tripletsPerBottomDublet, nParallelMiddleSPs, maxMBDublets,
      middleIndexOffset, bottomDubletIndex));
  const unsigned int tripletIndexRow = atomicAdd(tripletIndexRowPtr, 1);
  assert(tripletIndexRow < maxMTDublets);
  const unsigned int tripletIndex = atomicAdd(tripletCount, 1);
  assert(tripletIndex < maxTriplets);

  // Collect the maximal value of tripletIndexRow + 1 (since we want the
  // count, not the index values) for the next kernel.
  atomicMax(maxTripletsPerSpB, tripletIndexRow + 1);

  // Save the index of the triplet candidate, which will be created now.
  ACTS_CUDA_MATRIX3D_ELEMENT(tripletIndices, nParallelMiddleSPs, maxMBDublets,
                             maxMTDublets, middleIndexOffset, bottomDubletIndex,
                             tripletIndexRow) = tripletIndex;

  // Now store the triplet in the above mentioned location.
  Details::Triplet triplet = {bottomIndex,   middleIndex,
                              topIndex,      Im,
                              B / sqrtf(S2), -(Im * impactWeightFactor)};
  triplets[tripletIndex] = triplet;

  return;
}

/// Kernel performing the "2 fixed spacepoint filtering" of the triplets
///
/// @param[in] seedWeight Pointer to the user-provided seed weight calculating
///            function
/// @param[in] singleSeedCut Pointer to the user-provided seed filtering
///            function
/// @param[in] middleIndexStart The middle spacepoint index that the kernel was
///            "started from"
/// @param[in] maxMBDublets The maximal number of middle-bottom dublets found
///            for any middle spacepoint
/// @param[in] maxMTDublets The maximal number of middle-top dublets found for
///            any middle spacepoint
/// @param[in] maxTriplets The maximum number of triplets for which memory is
///            booked
/// @param[in] nAllTriplets The number of triplets that were reconstructed for
///            this middle spacepoint group
/// @param[in] nParallelMiddleSPs The number of middle spacepoints that the
///            "largest" kernels may be started on in parallel
/// @param[in] nMiddleSPsProcessed The number of middle spacepoints that the
///            kernel was started on in parallel
/// @param[in] middleBottomCounts 1-D array of the number of middle-bottom
///            dublets found for each middle spacepoint
/// @param[in] nBottomSPs The number of bottom spacepoints in @c bottomSPs
/// @param[in] bottomSPs Properties of all of the bottom spacepoints
/// @param[in] nMiddleSPs The number of middle spacepoints in @c middleSPs
/// @param[in] middleSPs Properties of all of the middle spacepoints
/// @param[in] nTopSPs The number of top spacepoints in @c topSPs
/// @param[in] topSPs Properties of all of the top spacepoints
/// @param[in] tripletsPerBottomDublet 1-dimensional array of the triplet
///            counts for each bottom spacepoint
/// @param[in] tripletIndices 2-dimensional matrix of the indices of the
///            triplets created for each middle-bottom spacepoint dublet
/// @param[in] allTriplets 1-dimensional array of all the found triplets
/// @param[in] deltaInvHelixDiameter Parameter from @c Acts::SeedFilterConfig
/// @param[in] deltaRMin Parameter from @c Acts::SeedFilterConfig
/// @param[in] compatSeedWeight Parameter from @c Acts::SeedFilterConfig
/// @param[in] compatSeedLimit Parameter from @c Acts::SeedFilterConfig
/// @param[out] nFilteredTriplets Pointer to the scalar counting all triplets
///             that survive this filter
/// @param[out] filteredTriplets 1-dimensional array of triplets that survive
///             this filter
///
__global__ void filterTriplets2Sp(
    TripletFilterConfig::seedWeightFunc_t seedWeight,
    TripletFilterConfig::singleSeedCutFunc_t singleSeedCut,
    std::size_t middleIndexStart, unsigned int maxMBDublets,
    unsigned int maxMTDublets, unsigned int maxTriplets,
    unsigned int nAllTriplets, std::size_t nParallelMiddleSPs,
    std::size_t nMiddleSPsProcessed, unsigned int* middleBottomCounts,
    std::size_t nBottomSPs, const Details::SpacePoint* bottomSPs,
    std::size_t nMiddleSPs, const Details::SpacePoint* middleSPs,
    std::size_t nTopSPs, const Details::SpacePoint* topSPs,
    const unsigned int* tripletsPerBottomDublet,
    const std::size_t* tripletIndices, const Details::Triplet* allTriplets,
    float deltaInvHelixDiameter, float deltaRMin, float compatSeedWeight,
    std::size_t compatSeedLimit, unsigned int* nFilteredTriplets,
    Details::Triplet* filteredTriplets) {
  // Sanity checks.
  assert(seedWeight != nullptr);
  assert(singleSeedCut != nullptr);
  assert(middleIndexStart + nMiddleSPsProcessed <= nMiddleSPs);

  // Find the middle spacepoint index to operate on.
  const unsigned int middleIndexOffset = blockIdx.x * blockDim.x + threadIdx.x;
  if (middleIndexOffset >= nMiddleSPsProcessed) {
    return;
  }
  const unsigned int middleIndex = middleIndexStart + middleIndexOffset;
  assert(middleIndex < nMiddleSPs);

  // Find the middle-bottom dublet to operate on.
  const unsigned int middleBottomPairCount = middleBottomCounts[middleIndex];
  const unsigned int bottomDubletIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if (bottomDubletIndex >= middleBottomPairCount) {
    return;
  }

  // Find the triplet to operate on.
  const unsigned int nTripletsForMiddleBottom = ACTS_CUDA_MATRIX2D_ELEMENT(
      tripletsPerBottomDublet, nParallelMiddleSPs, maxMBDublets,
      middleIndexOffset, bottomDubletIndex);
  const unsigned int tripletCandidateIndex =
      blockIdx.z * blockDim.z + threadIdx.z;
  if (tripletCandidateIndex >= nTripletsForMiddleBottom) {
    return;
  }

  // Get the index of this triplet.
  const std::size_t triplet1Index = ACTS_CUDA_MATRIX3D_ELEMENT(
      tripletIndices, nParallelMiddleSPs, maxMBDublets, maxMTDublets,
      middleIndexOffset, bottomDubletIndex, tripletCandidateIndex);
  assert(triplet1Index < nAllTriplets);

  // Load this triplet into the thread.
  Details::Triplet triplet1 = allTriplets[triplet1Index];

  // Pre-compute some variables.
  float lowerLimitCurv = triplet1.invHelixDiameter - deltaInvHelixDiameter;
  float upperLimitCurv = triplet1.invHelixDiameter + deltaInvHelixDiameter;
  float currentTop_r = topSPs[triplet1.topIndex].radius;

  // Allow only a maximum number of top spacepoints in the filtering. Since a
  // limit is coming from @c compatSeedLimit anyway, this could potentially be
  // re-written with an array allocation, instead of statically defining the
  // array's size.
  static constexpr std::size_t MAX_TOP_SP = 10;
  assert(compatSeedLimit < MAX_TOP_SP);
  float compatibleSeedR[MAX_TOP_SP];
  std::size_t nCompatibleSeedR = 0;

  // Loop over all the other triplets found for this bottom-middle dublet.
  for (std::size_t i = 0; i < nTripletsForMiddleBottom; ++i) {
    // Don't consider the same triplet that the thread is evaluating in the
    // first place.
    if (i == tripletCandidateIndex) {
      continue;
    }
    // Get the index of the second triplet.
    const std::size_t triplet2Index = ACTS_CUDA_MATRIX3D_ELEMENT(
        tripletIndices, nParallelMiddleSPs, maxMBDublets, maxMTDublets,
        middleIndexOffset, bottomDubletIndex, i);
    assert(triplet2Index < nAllTriplets);
    assert(triplet2Index != triplet1Index);

    // Load the second triplet into the thread.
    const Details::Triplet triplet2 = allTriplets[triplet2Index];
    assert(triplet1.bottomIndex == triplet2.bottomIndex);

    // compared top SP should have at least deltaRMin distance
    float otherTop_r = topSPs[triplet2.topIndex].radius;
    float deltaR = currentTop_r - otherTop_r;
    if (fabs(deltaR) < deltaRMin) {
      continue;
    }

    // curvature difference within limits?
    // TODO: how much slower than sorting all vectors by curvature
    // and breaking out of loop? i.e. is vector size large (e.g. in jets?)
    if (triplet2.invHelixDiameter < lowerLimitCurv) {
      continue;
    }
    if (triplet2.invHelixDiameter > upperLimitCurv) {
      continue;
    }

    bool newCompSeed = true;
    for (std::size_t k = 0; k < nCompatibleSeedR; ++k) {
      // original ATLAS code uses higher min distance for 2nd found compatible
      // seed (20mm instead of 5mm)
      // add new compatible seed only if distance larger than rmin to all
      // other compatible seeds
      if (fabs(compatibleSeedR[k] - otherTop_r) < deltaRMin) {
        newCompSeed = false;
        break;
      }
    }
    if (newCompSeed) {
      compatibleSeedR[nCompatibleSeedR++] = otherTop_r;
      assert(nCompatibleSeedR < MAX_TOP_SP);
      triplet1.weight += compatSeedWeight;
    }
    if (nCompatibleSeedR >= compatSeedLimit) {
      break;
    }
  }

  // Decide whether to keep the triplet or not.
  triplet1.weight +=
      seedWeight(bottomSPs[triplet1.bottomIndex], middleSPs[middleIndex],
                 topSPs[triplet1.topIndex]);
  if (!singleSeedCut(triplet1.weight, bottomSPs[triplet1.bottomIndex],
                     middleSPs[middleIndex], topSPs[triplet1.topIndex])) {
    return;
  }

  // Put the triplet into the "filtered list".
  const unsigned int tripletRow = atomicAdd(nFilteredTriplets, 1);
  assert(tripletRow < nAllTriplets);
  filteredTriplets[tripletRow] = triplet1;
  return;
}

}  // namespace Kernels

namespace Details {

std::vector<std::vector<Triplet>> findTriplets(
    const Info::Device& device, std::size_t maxBlockSize,
    const DubletCounts& dubletCounts, const SeedFilterConfig& seedConfig,
    const TripletFilterConfig& filterConfig, std::size_t nBottomSPs,
    const device_array<SpacePoint>& bottomSPs, std::size_t nMiddleSPs,
    const device_array<SpacePoint>& middleSPs, std::size_t nTopSPs,
    const device_array<SpacePoint>& topSPs,
    const device_array<unsigned int>& middleBottomCounts,
    const device_array<std::size_t>& middleBottomDublets,
    const device_array<unsigned int>& middleTopCounts,
    const device_array<std::size_t>& middleTopDublets,
    float maxScatteringAngle2, float sigmaScattering, float minHelixDiameter2,
    float pT2perRadius, float impactMax) {
  // Calculate the parallelisation for the parameter transformation.
  const int numBlocksLT =
      (dubletCounts.nDublets + maxBlockSize - 1) / maxBlockSize;

  // Create the arrays holding the linear transformed spacepoint parameters.
  auto bottomSPLinTransArray =
      make_device_array<LinCircle>(nMiddleSPs * dubletCounts.maxMBDublets);
  auto topSPLinTransArray =
      make_device_array<LinCircle>(nMiddleSPs * dubletCounts.maxMTDublets);

  // Launch the coordinate transformations.
  Kernels::transformCoordinates<<<numBlocksLT, maxBlockSize>>>(
      dubletCounts.nDublets, dubletCounts.maxMBDublets,
      dubletCounts.maxMTDublets, nBottomSPs, bottomSPs.get(), nMiddleSPs,
      middleSPs.get(), nTopSPs, topSPs.get(), middleBottomCounts.get(),
      middleBottomDublets.get(), middleTopCounts.get(), middleTopDublets.get(),
      bottomSPLinTransArray.get(), topSPLinTransArray.get());
  ACTS_CUDA_ERROR_CHECK(hipGetLastError());
  ACTS_CUDA_ERROR_CHECK(hipDeviceSynchronize());

  // With the information from @c Acts::Cuda::Details::DubletCounts, figure out
  // how many middle spacepoints we could handle at the same time in the triplet
  // finding/filtering.

  // For one middle spacepoint we need the following amount:
  const std::size_t memorySizePerMiddleSP =
      // First let's consider the storage of the triplet objects themselves.
      2 * dubletCounts.maxTriplets * sizeof(Triplet) +
      // Then the objects holding indices to the triplets per middle-bottom
      // dublet.
      dubletCounts.maxMBDublets * sizeof(unsigned int) +
      dubletCounts.maxMBDublets * dubletCounts.maxMTDublets *
          sizeof(std::size_t) +
      // Finally the array holding the filtered triplet counts per middle
      // spacepoint.
      sizeof(unsigned int);

  // See how many we can fit into the (still) available memory.
  const std::size_t nParallelMiddleSPs =
      std::min(MemoryManager::instance().availableMemory(device.id) /
                   memorySizePerMiddleSP,
               nMiddleSPs);
  assert(nParallelMiddleSPs > 0);

  // Helper variables for handling the various object counts in device memory.
  enum ObjectCountType : int {
    AllTriplets = 0,        ///< All viable triplets
    FilteredTriplets = 1,   ///< Triplets after the "2SpFixed" filtering
    MaxTripletsPerSpB = 2,  ///< Maximal number of triplets found per SpB
    NObjectCountTypes = 3   ///< The number of different object/counter types
  };

  // Set up the object counters in device memory. The host array is only used to
  // reset the device memory before every iteration.
  auto objectCountsHostNull = make_host_array<unsigned int>(NObjectCountTypes);
  memset(objectCountsHostNull.get(), 0,
         NObjectCountTypes * sizeof(unsigned int));
  auto objectCountsHost = make_host_array<unsigned int>(NObjectCountTypes);
  auto objectCounts = make_device_array<unsigned int>(NObjectCountTypes);

  // Allocate enough memory for triplet candidates that would suffice for every
  // middle spacepoint.
  auto allTriplets =
      make_device_array<Triplet>(nParallelMiddleSPs * dubletCounts.maxTriplets);
  auto filteredTriplets =
      make_device_array<Triplet>(nParallelMiddleSPs * dubletCounts.maxTriplets);
  auto filteredTripletsHost =
      make_host_array<Triplet>(nParallelMiddleSPs * dubletCounts.maxTriplets);

  // Allocate and initialise the array holding the per bottom dublet triplet
  // numbers.
  auto tripletsPerBottomDubletHost = make_host_array<unsigned int>(
      nParallelMiddleSPs * dubletCounts.maxMBDublets);
  memset(tripletsPerBottomDubletHost.get(), 0,
         nParallelMiddleSPs * dubletCounts.maxMBDublets * sizeof(unsigned int));
  auto tripletsPerBottomDublet = make_device_array<unsigned int>(
      nParallelMiddleSPs * dubletCounts.maxMBDublets);

  // Allocate the array holding the indices of the triplets found for a given
  // bottom-middle spacepoint combination.
  auto tripletIndices = make_device_array<std::size_t>(
      nParallelMiddleSPs * dubletCounts.maxMBDublets *
      dubletCounts.maxMTDublets);

  // Allocate and initialise the arrays holding the per-middle-spacepoint
  // filtered triplet counts.
  auto filteredTripletCountsHostNull =
      make_host_array<unsigned int>(nParallelMiddleSPs);
  memset(filteredTripletCountsHostNull.get(), 0,
         nParallelMiddleSPs * sizeof(unsigned int));
  auto filteredTripletCountsHost =
      make_host_array<unsigned int>(nParallelMiddleSPs);
  auto filteredTripletCounts =
      make_device_array<unsigned int>(nParallelMiddleSPs);

  // Block size used in the triplet finding.
  const std::size_t blockSize = std::sqrt(maxBlockSize);

  // Create the result object.
  std::vector<std::vector<Triplet>> result(nMiddleSPs);

  // Copy the dublet counts back to the host.
  auto middleBottomCountsHost = make_host_array<unsigned int>(nMiddleSPs);
  copyToHost(middleBottomCountsHost, middleBottomCounts, nMiddleSPs);
  auto middleTopCountsHost = make_host_array<unsigned int>(nMiddleSPs);
  copyToHost(middleTopCountsHost, middleTopCounts, nMiddleSPs);

  // Execute the triplet finding and filtering in the maximal allowed groups of
  // middle spacepoints.
  for (std::size_t middleIndex = 0; middleIndex < nMiddleSPs;
       middleIndex += nParallelMiddleSPs) {
    // Reset the device arrays.
    copyToDevice(objectCounts, objectCountsHostNull, NObjectCountTypes);
    copyToDevice(tripletsPerBottomDublet, tripletsPerBottomDubletHost,
                 nParallelMiddleSPs * dubletCounts.maxMBDublets);

    // The number of middle spacepoints to process in this iteration.
    const std::size_t nMiddleSPsProcessed =
        std::min(nParallelMiddleSPs, nMiddleSPs - middleIndex);

    // Calculate the parallelisation for the triplet finding for this collection
    // of middle spacepoints.
    const dim3 blockSizeFT(1, maxBlockSize);
    const dim3 numBlocksFT(
        (nMiddleSPsProcessed + blockSizeFT.x - 1) / blockSizeFT.x,
        (dubletCounts.maxTriplets + blockSizeFT.y - 1) / blockSizeFT.y);
    assert(dubletCounts.maxTriplets > 0);

    // Launch the triplet finding for this middle spacepoint.
    Kernels::findTriplets<<<numBlocksFT, blockSizeFT>>>(
        // Parameters needed to use all the arrays.
        middleIndex, dubletCounts.maxMBDublets, dubletCounts.maxMTDublets,
        dubletCounts.maxTriplets, nParallelMiddleSPs, nMiddleSPsProcessed,
        // Parameters of all of the spacepoints.
        nBottomSPs, bottomSPs.get(), nMiddleSPs, middleSPs.get(), nTopSPs,
        topSPs.get(),
        // Arrays describing the identified dublets.
        middleBottomCounts.get(), middleBottomDublets.get(),
        middleTopCounts.get(), middleTopDublets.get(),
        // The transformed parameters of the bottom and top spacepoints for
        // spacepoints taking part in dublets.
        bottomSPLinTransArray.get(), topSPLinTransArray.get(),
        // Configuration constants.
        maxScatteringAngle2, sigmaScattering, minHelixDiameter2, pT2perRadius,
        impactMax, seedConfig.impactWeightFactor,
        // Variables storing the results of the triplet finding.
        tripletsPerBottomDublet.get(), tripletIndices.get(),
        objectCounts.get() + MaxTripletsPerSpB,
        objectCounts.get() + AllTriplets, allTriplets.get());
    ACTS_CUDA_ERROR_CHECK(hipGetLastError());
    ACTS_CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Retrieve the object counts.
    copyToHost(objectCountsHost, objectCounts, NObjectCountTypes);
    const unsigned int nAllTriplets = objectCountsHost.get()[AllTriplets];
    const unsigned int nMaxTripletsPerSpB =
        objectCountsHost.get()[MaxTripletsPerSpB];

    // If no triplet has been found, stop here for this middle spacepoint range.
    if (nAllTriplets == 0) {
      continue;
    }

    // Calculate the parallelisation for the "2SpFixed" filtering of the
    // triplets.
    const dim3 blockSizeF2SP(1, blockSize, blockSize);
    const dim3 numBlocksF2SP(
        (nMiddleSPsProcessed + blockSizeF2SP.x - 1) / blockSizeF2SP.x,
        (dubletCounts.maxMBDublets + blockSizeF2SP.y - 1) / blockSizeF2SP.y,
        (nMaxTripletsPerSpB + blockSizeF2SP.z - 1) / blockSizeF2SP.z);
    assert(dubletCounts.maxMBDublets > 0);
    assert(nMaxTripletsPerSpB > 0);

    // Launch the "2SpFixed" filtering of the triplets.
    assert(filterConfig.seedWeight != nullptr);
    assert(filterConfig.singleSeedCut != nullptr);
    Kernels::filterTriplets2Sp<<<numBlocksF2SP, blockSizeF2SP>>>(
        // Pointers to the user provided filter functions.
        filterConfig.seedWeight, filterConfig.singleSeedCut,
        // Parameters needed to use all the arrays.
        middleIndex, dubletCounts.maxMBDublets, dubletCounts.maxMTDublets,
        dubletCounts.maxTriplets, nAllTriplets, nParallelMiddleSPs,
        nMiddleSPsProcessed, middleBottomCounts.get(),
        // Parameters of all of the spacepoints.
        nBottomSPs, bottomSPs.get(), nMiddleSPs, middleSPs.get(), nTopSPs,
        topSPs.get(),
        // Variables holding the results of the triplet finding.
        tripletsPerBottomDublet.get(), tripletIndices.get(), allTriplets.get(),
        // Configuration constants.
        seedConfig.deltaInvHelixDiameter, seedConfig.deltaRMin,
        seedConfig.compatSeedWeight, seedConfig.compatSeedLimit,
        // Variables storing the results of the filtering.
        objectCounts.get() + FilteredTriplets, filteredTriplets.get());
    ACTS_CUDA_ERROR_CHECK(hipGetLastError());
    ACTS_CUDA_ERROR_CHECK(hipDeviceSynchronize());

    // Retrieve the result counts of the filtering.
    copyToHost(objectCountsHost, objectCounts, NObjectCountTypes);

    // The number of triplets that survived the 2Sp filtering.
    const unsigned int nFilteredTriplets =
        objectCountsHost.get()[FilteredTriplets];
    if (nFilteredTriplets == 0) {
      continue;
    }

    // Move the filtered triplets back to the host for the final selection.
    ACTS_CUDA_ERROR_CHECK(hipMemcpy(
        filteredTripletsHost.get(), filteredTriplets.get(),
        nFilteredTriplets * sizeof(Triplet), hipMemcpyDeviceToHost));

    // Fill the output variable.
    for (std::size_t i = 0; i < nFilteredTriplets; ++i) {
      // Access the triplet.
      const Triplet& triplet = filteredTripletsHost.get()[i];
      // Put it into the output object.
      result[triplet.middleIndex].push_back(triplet);
    }
  }

  // Return the indices of all identified triplets.
  assert(result.size() == nMiddleSPs);
  return result;
}

}  // namespace Details
}  // namespace Cuda
}  // namespace Acts
