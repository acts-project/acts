// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

// CUDA plugin include(s).
#include "Acts/Plugins/Cuda/Utilities/StreamWrapper.hpp"

#include "ErrorCheck.cuh"
#include "StreamHandlers.cuh"

// CUDA include(s).
#include <hip/hip_runtime.h>

namespace Acts {
namespace Cuda {

StreamWrapper::StreamWrapper(void* stream, bool ownsStream)
    : m_stream(stream), m_ownsStream(ownsStream) {}

StreamWrapper::StreamWrapper(StreamWrapper&& parent)
    : m_stream(parent.m_stream), m_ownsStream(parent.m_ownsStream) {
  parent.m_stream = nullptr;
  parent.m_ownsStream = false;
}

StreamWrapper::~StreamWrapper() {
  // Destroy the stream, if we still hold it.
  if (m_stream && m_ownsStream) {
    ACTS_CUDA_ERROR_CHECK(hipStreamDestroy(getStreamFrom(*this)));
  }
}

StreamWrapper& StreamWrapper::operator=(StreamWrapper&& rhs) {
  // Check whether anything needs to be done.
  if (this == &rhs) {
    return *this;
  }

  // Destroy the current stream, if we hold one.
  if (m_stream && m_ownsStream) {
    ACTS_CUDA_ERROR_CHECK(hipStreamDestroy(getStreamFrom(*this)));
  }

  // Perform the move.
  m_stream = rhs.m_stream;
  m_ownsStream = rhs.m_ownsStream;
  rhs.m_stream = nullptr;
  rhs.m_ownsStream = false;

  // Return this object.
  return *this;
}

void StreamWrapper::synchronize() const {
  // Use CUDA to wait for all tasks to finish in the stream.
  ACTS_CUDA_ERROR_CHECK(hipStreamSynchronize(getStreamFrom(*this)));
  return;
}

StreamWrapper createStreamFor(const Acts::Cuda::Info::Device& device) {
  // Create the stream for the selected device.
  ACTS_CUDA_ERROR_CHECK(hipSetDevice(device.id));
  hipStream_t stream = nullptr;
  ACTS_CUDA_ERROR_CHECK(
      hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // Return the new object.
  return StreamWrapper(stream);
}

}  // namespace Cuda
}  // namespace Acts
