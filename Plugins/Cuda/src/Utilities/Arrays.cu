// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

// CUDA plugin include(s).
#include "Acts/Plugins/Cuda/Seeding2/Details/Types.hpp"
#include "Acts/Plugins/Cuda/Utilities/Arrays.hpp"
#include "Acts/Plugins/Cuda/Utilities/MemoryManager.hpp"

#include "ErrorCheck.cuh"
#include "StreamHandlers.cuh"

// CUDA include(s).
#include <hip/hip_runtime.h>

// System include(s).
#include <cstdlib>

namespace Acts {
namespace Cuda {
namespace Details {

void DeviceArrayDeleter::operator()(void*) {
  // The memory is managed by @c Acts::Cuda::MemoryManager, don't do anything
  // here.
  return;
}

void HostArrayDeleter::operator()(void* ptr) {
  // Ignore null-pointers.
  if (ptr == nullptr) {
    return;
  }

  // Free the host memory.
  free(ptr);
  return;
}

}  // namespace Details

template <typename T>
device_array<T> make_device_array(std::size_t size) {
  // Allocate the memory.
  T* ptr = nullptr;
  if (size != 0) {
    ptr = static_cast<T*>(MemoryManager::instance().allocate(size * sizeof(T)));
  }
  // Create the smart pointer.
  return device_array<T>(ptr);
}

template <typename T>
host_array<T> make_host_array(std::size_t size) {
  // Allocate the memory.
  T* ptr = nullptr;
  if (size != 0) {
    ptr = static_cast<T*>(malloc(size * sizeof(T)));
  }
  // Create the smart pointer.
  return host_array<T>(ptr);
}

template <typename T>
void copyToDevice(device_array<T>& dev, const host_array<T>& host,
                  std::size_t arraySize) {
  ACTS_CUDA_ERROR_CHECK(hipMemcpy(dev.get(), host.get(), arraySize * sizeof(T),
                                   hipMemcpyHostToDevice));
  return;
}

template <typename T>
void copyToDevice(device_array<T>& dev, const host_array<T>& host,
                  std::size_t arraySize, const StreamWrapper& stream) {
  ACTS_CUDA_ERROR_CHECK(
      hipMemcpyAsync(dev.get(), host.get(), arraySize * sizeof(T),
                      hipMemcpyHostToDevice, getStreamFrom(stream)));
  return;
}

template <typename T>
void copyToHost(host_array<T>& host, const device_array<T>& dev,
                std::size_t arraySize) {
  ACTS_CUDA_ERROR_CHECK(hipMemcpy(host.get(), dev.get(), arraySize * sizeof(T),
                                   hipMemcpyDeviceToHost));
  return;
}

template <typename T>
void copyToHost(host_array<T>& host, const device_array<T>& dev,
                std::size_t arraySize, const StreamWrapper& stream) {
  ACTS_CUDA_ERROR_CHECK(
      hipMemcpyAsync(host.get(), dev.get(), arraySize * sizeof(T),
                      hipMemcpyDeviceToHost, getStreamFrom(stream)));
  return;
}

}  // namespace Cuda
}  // namespace Acts

/// Helper macro for instantiating the template code for a given type
///
/// Note that nvcc (at least as of CUDA version 11.0.2) does not allow us to
/// instantiate our custom unique pointer types through their typedef'd names.
/// That's why the following expressions are as long as they are.
///
#define INST_ARRAY_FOR_TYPE(TYPE)                                              \
  template class std::unique_ptr<TYPE,                                         \
                                 Acts::Cuda::Details::DeviceArrayDeleter>;     \
  template std::unique_ptr<TYPE, Acts::Cuda::Details::DeviceArrayDeleter>      \
      Acts::Cuda::make_device_array<TYPE>(std::size_t);                        \
  template class std::unique_ptr<TYPE, Acts::Cuda::Details::HostArrayDeleter>; \
  template std::unique_ptr<TYPE, Acts::Cuda::Details::HostArrayDeleter>        \
      Acts::Cuda::make_host_array<TYPE>(std::size_t);                          \
  template void Acts::Cuda::copyToDevice<TYPE>(                                \
      std::unique_ptr<TYPE, Acts::Cuda::Details::DeviceArrayDeleter>&,         \
      const std::unique_ptr<TYPE, Acts::Cuda::Details::HostArrayDeleter>&,     \
      std::size_t);                                                            \
  template void Acts::Cuda::copyToDevice<TYPE>(                                \
      std::unique_ptr<TYPE, Acts::Cuda::Details::DeviceArrayDeleter>&,         \
      const std::unique_ptr<TYPE, Acts::Cuda::Details::HostArrayDeleter>&,     \
      std::size_t, const Acts::Cuda::StreamWrapper&);                          \
  template void Acts::Cuda::copyToHost<TYPE>(                                  \
      std::unique_ptr<TYPE, Acts::Cuda::Details::HostArrayDeleter>&,           \
      const std::unique_ptr<TYPE, Acts::Cuda::Details::DeviceArrayDeleter>&,   \
      std::size_t);                                                            \
  template void Acts::Cuda::copyToHost<TYPE>(                                  \
      std::unique_ptr<TYPE, Acts::Cuda::Details::HostArrayDeleter>&,           \
      const std::unique_ptr<TYPE, Acts::Cuda::Details::DeviceArrayDeleter>&,   \
      std::size_t, const Acts::Cuda::StreamWrapper&)

// Instantiate the templated functions for all primitive types.
INST_ARRAY_FOR_TYPE(char);
INST_ARRAY_FOR_TYPE(unsigned char);
INST_ARRAY_FOR_TYPE(short);
INST_ARRAY_FOR_TYPE(unsigned short);
INST_ARRAY_FOR_TYPE(int);
INST_ARRAY_FOR_TYPE(unsigned int);
INST_ARRAY_FOR_TYPE(long);
INST_ARRAY_FOR_TYPE(unsigned long);
INST_ARRAY_FOR_TYPE(long long);
INST_ARRAY_FOR_TYPE(unsigned long long);
INST_ARRAY_FOR_TYPE(float);
INST_ARRAY_FOR_TYPE(double);

// Instantiate them for any necessary custom type(s) as well.
INST_ARRAY_FOR_TYPE(Acts::Cuda::Details::SpacePoint);
INST_ARRAY_FOR_TYPE(Acts::Cuda::Details::DubletCounts);
INST_ARRAY_FOR_TYPE(Acts::Cuda::Details::LinCircle);
INST_ARRAY_FOR_TYPE(Acts::Cuda::Details::Triplet);

// Clean up.
#undef INST_ARRAY_FOR_TYPE
