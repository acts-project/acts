// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include "Acts/Plugins/Cuda/Utilities/CpuVector.hpp"

#include <iostream>
#include <memory>

#include "CudaUtils.cu"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Acts {

template <typename var_t>
class CpuVector;

template <typename var_t>
class CudaVector {
 public:
  CudaVector() = delete;
  CudaVector(std::size_t size) {
    m_size = size;
    ACTS_CUDA_ERROR_CHECK(
        hipMalloc((var_t**)&m_devPtr, m_size * sizeof(var_t)));
  }

  CudaVector(std::size_t size, var_t* vector) {
    m_size = size;
    ACTS_CUDA_ERROR_CHECK(
        hipMalloc((var_t**)&m_devPtr, m_size * sizeof(var_t)));
    copyH2D(vector, m_size, 0);
  }

  CudaVector(std::size_t size, var_t* vector, std::size_t len,
             std::size_t offset) {
    m_size = size;
    ACTS_CUDA_ERROR_CHECK(
        hipMalloc((var_t**)&m_devPtr, m_size * sizeof(var_t)));
    copyH2D(vector, len, offset);
  }

  ~CudaVector() {
    if (m_devPtr)
      hipFree(m_devPtr);
  }

  var_t* get(std::size_t offset = 0) { return m_devPtr + offset; }

  void copyH2D(var_t* vector, std::size_t len, std::size_t offset) {
    ACTS_CUDA_ERROR_CHECK(hipMemcpy(m_devPtr + offset, vector,
                                     len * sizeof(var_t),
                                     hipMemcpyHostToDevice));
  }
  void copyH2D(var_t* vector, std::size_t len, std::size_t offset,
               hipStream_t* stream) {
    ACTS_CUDA_ERROR_CHECK(hipMemcpyAsync(m_devPtr + offset, vector,
                                          len * sizeof(var_t),
                                          hipMemcpyHostToDevice, *stream));
  }

  void zeros() {
    ACTS_CUDA_ERROR_CHECK(hipMemset(m_devPtr, 0, m_size * sizeof(var_t)));
  }

 private:
  var_t* m_devPtr = nullptr;
  std::size_t m_size;
};
}  // namespace Acts
