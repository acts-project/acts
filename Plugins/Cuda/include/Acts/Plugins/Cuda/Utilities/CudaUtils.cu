// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#pragma once

#include <iostream>

#include <hip/hip_runtime.h>


#define ACTS_CUDA_ERROR_CHECK(ans) \
  { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line,
                       bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDAassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}
