// This file is part of the Acts project.
//
// Copyright (C) 2020 CERN for the benefit of the Acts project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at http://mozilla.org/MPL/2.0/.

#pragma once

#include "Acts/Plugins/Cuda/Utilities/CpuMatrix.hpp"

#include <iostream>
#include <memory>

#include "CudaUtils.cu"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace Acts {

template <typename var_t>
class UsmMatrix {
 public:
  UsmMatrix() = delete;
  UsmMatrix(std::size_t nRows, std::size_t nCols) {
    m_setSize(nRows, nCols);
    ACTS_CUDA_ERROR_CHECK(hipMallocManaged((var_t**)&m_devPtr,
                                            m_nRows * m_nCols * sizeof(var_t)));
    hipDeviceSynchronize();
  }

  ~UsmMatrix() {
    hipDeviceSynchronize();
    hipFree(m_devPtr);
  }

  var_t* get(std::size_t row = 0, std::size_t col = 0) {
    int offset = row + col * m_nRows;
    return m_devPtr + offset;
  }

  void set(std::size_t row, std::size_t col, var_t val) {
    std::size_t offset = row + col * m_nRows;
    m_devPtr[offset] = val;
  }

  void zeros() { memset(m_devPtr, 0, m_size * sizeof(var_t)); }

 private:
  var_t* m_devPtr;
  std::size_t m_nCols;
  std::size_t m_nRows;
  std::size_t m_size;

  void m_setSize(std::size_t row, std::size_t col) {
    m_nRows = row;
    m_nCols = col;
    m_size = m_nRows * m_nCols;
  }
};

}  // namespace Acts
