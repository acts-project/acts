#include "hip/hip_runtime.h"
// This file is part of the ACTS project.
//
// Copyright (C) 2016 CERN for the benefit of the ACTS project
//
// This Source Code Form is subject to the terms of the Mozilla Public
// License, v. 2.0. If a copy of the MPL was not distributed with this
// file, You can obtain one at https://mozilla.org/MPL/2.0/.

#include <boost/test/unit_test.hpp>

#include <Acts/Plugins/ExaTrkX/detail/ConnectedComponents.cuh>

#include <filesystem>
#include <fstream>
#include <random>
#include <set>
#include <vector>

#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/connected_components.hpp>

using namespace Acts::detail;

using BoostGraph =
    boost::adjacency_list<boost::vecS, boost::vecS, boost::undirectedS>;

using Vi = std::vector<int>;

Vi checkLabeling(const std::vector<int> &src, const std::vector<int> &tgt) {
  std::size_t numNodes = std::max(*std::max_element(src.begin(), src.end()),
                                  *std::max_element(tgt.begin(), tgt.end())) +
                         1;

  int *cudaSrc, *cudaTgt;
  hipMalloc(&cudaSrc, src.size() * sizeof(int));
  hipMalloc(&cudaTgt, tgt.size() * sizeof(int));
  hipMemcpy(cudaSrc, src.data(), src.size() * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(cudaTgt, tgt.data(), src.size() * sizeof(int),
             hipMemcpyHostToDevice);

  int *cudaLabels;
  hipMalloc(&cudaLabels, numNodes * sizeof(int));
  int *cudaLabelsNext;
  hipMalloc(&cudaLabelsNext, numNodes * sizeof(int));

  labelConnectedComponents<<<1, 1024>>>(src.size(), cudaSrc, cudaTgt, numNodes,
                                        cudaLabels, cudaLabelsNext);

  std::vector<int> labelsFromCuda(numNodes);
  hipMemcpy(labelsFromCuda.data(), cudaLabels, numNodes * sizeof(int),
             hipMemcpyDeviceToHost);

  BoostGraph G(numNodes);

  for (int i = 0; i < src.size(); ++i) {
    boost::add_edge(src[i], tgt[i], G);
  }

  std::vector<std::size_t> cpuLabels(numNodes);
  boost::connected_components(G, &cpuLabels[0]);

  // print
  std::cout << "cpu labels:     ";
  for (int i = 0; i < numNodes; ++i) {
    std::cout << cpuLabels[i] << " ";
  }
  std::cout << std::endl;

  std::cout << "my CUDA labels: ";
  for (int i = 0; i < numNodes; ++i) {
    std::cout << labelsFromCuda[i] << " ";
  }
  std::cout << std::endl;

  // check systematically
  std::map<int, int> boostToCuda;
  for (int i = 0; i < numNodes; ++i) {
    if (boostToCuda.contains(cpuLabels[i])) {
      BOOST_CHECK_EQUAL(labelsFromCuda[i], boostToCuda.at(cpuLabels[i]));
    } else {
      auto [it, success] =
          boostToCuda.insert({cpuLabels[i], labelsFromCuda[i]});
      BOOST_CHECK(success);
    }
  }

  return labelsFromCuda;
}

BOOST_AUTO_TEST_CASE(simple_test_1) {
  Vi src{0, 1, 2, 3};
  Vi tgt{1, 2, 3, 4};
  checkLabeling(src, tgt);
}

BOOST_AUTO_TEST_CASE(simple_test_2) {
  Vi src{0, 1, 2, 4, 5, 6};
  Vi tgt{1, 2, 3, 5, 6, 7};
  checkLabeling(src, tgt);
}

BOOST_AUTO_TEST_CASE(simple_test_3) {
  Vi src{4, 3, 2, 1};
  Vi tgt{3, 2, 1, 0};
  checkLabeling(src, tgt);
}

void testRelabeling(const Vi &labels, const Vi &refLabelMask,
                    const Vi &refPrefixSum, const Vi &refLabels) {
  dim3 blockDim = 32;
  dim3 gridDim = (labels.size() + blockDim.x - 1) / blockDim.x;

  // Copy labels to device
  int *cudaLabels;
  hipMalloc(&cudaLabels, labels.size() * sizeof(int));
  hipMemcpy(cudaLabels, labels.data(), labels.size() * sizeof(int),
             hipMemcpyHostToDevice);

  // Init label mask
  int *cudaLabelMask;
  hipMalloc(&cudaLabelMask, labels.size() * sizeof(int));
  hipMemset(cudaLabelMask, 0, labels.size() * sizeof(int));

  makeLabelMask<<<1, 256>>>(labels.size(), cudaLabels, cudaLabelMask);
  hipDeviceSynchronize();

  std::vector<int> labelMask(labels.size());
  hipMemcpy(labelMask.data(), cudaLabelMask, labelMask.size() * sizeof(int),
             hipMemcpyDeviceToHost);

  BOOST_CHECK_EQUAL_COLLECTIONS(labelMask.begin(), labelMask.end(),
                                refLabelMask.begin(), refLabelMask.end());

  // Prefix sum
  int *cudaPrefixSum;
  hipMalloc(&cudaPrefixSum, labels.size() * sizeof(int));
  thrust::exclusive_scan(thrust::device.on(0), cudaLabelMask,
                         cudaLabelMask + labels.size(), cudaPrefixSum);

  Vi prefixSum(labels.size());
  hipMemcpy(prefixSum.data(), cudaPrefixSum, labels.size() * sizeof(int),
             hipMemcpyDeviceToHost);
  BOOST_CHECK_EQUAL_COLLECTIONS(prefixSum.begin(), prefixSum.end(),
                                refPrefixSum.begin(), refPrefixSum.end());

  // Relabel
  mapEdgeLabels<<<1, 256>>>(labels.size(), cudaLabels, cudaPrefixSum);
  hipDeviceSynchronize();

  std::vector<int> labelsFromCuda(labels.size());
  hipMemcpy(labelsFromCuda.data(), cudaLabels, labels.size() * sizeof(int),
             hipMemcpyDeviceToHost);

  BOOST_CHECK_EQUAL_COLLECTIONS(labelsFromCuda.begin(), labelsFromCuda.end(),
                                refLabels.begin(), refLabels.end());
}

BOOST_AUTO_TEST_CASE(test_relabeling) {
  // clang-format off
  Vi labels      {0, 3, 5, 3, 0, 0};
  Vi refLabelMask{1, 0, 0, 1, 0, 1};
  Vi refPrefixSum{0, 1, 1, 1, 2, 2};
  Vi refLabels   {0, 1, 2, 1, 0, 0};
  // clang-format on

  testRelabeling(labels, refLabelMask, refPrefixSum, refLabels);
}

BOOST_AUTO_TEST_CASE(test_relabeling_2) {
  // clang-format off
  Vi labels      {1, 3, 5, 3, 1, 1};
  Vi refLabelMask{0, 1, 0, 1, 0, 1};
  Vi refPrefixSum{0, 0, 1, 1, 2, 2};
  Vi refLabels   {0, 1, 2, 1, 0, 0};
  // clang-format on

  testRelabeling(labels, refLabelMask, refPrefixSum, refLabels);
}

auto makeRandomGraph(std::size_t nodes, std::size_t edges) {
  std::default_random_engine rng(2345);
  std::uniform_int_distribution<> dist(0, nodes);
  std::set<std::pair<int, int>> set;
  Vi src(edges), tgt(edges);
  for (int n = 0; n < edges; ++n) {
    auto a = dist(rng);
    auto b = dist(rng);
    if (a == b) {
      continue;
    }
    auto s = std::min(a, b);
    auto t = std::max(a, b);
    auto [it, success] = set.insert({s, t});
    if (success) {
      src.at(n) = s;
      tgt.at(n) = t;
    }
  }

  return std::make_pair(src, tgt);
}

BOOST_AUTO_TEST_CASE(test_random_graph) {
  auto [src, tgt] = makeRandomGraph(5, 10);
  checkLabeling(src, tgt);
}

void testFullConnectedComponents(const Vi &src, const Vi &tgt) {
  const auto nNodes = std::max(*std::max_element(src.begin(), src.end()),
                               *std::max_element(tgt.begin(), tgt.end())) +
                      1;
  const auto nEdges = src.size();

  // print src and tgt
  /*
    std::cout << "src: ";
    for (int i = 0; i < src.size(); ++i) {
      std::cout << src[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "tgt: ";
    for (int i = 0; i < tgt.size(); ++i) {
      std::cout << tgt[i] << " ";
    }
    std::cout << std::endl;
  */
  hipStream_t stream;
  BOOST_REQUIRE_EQUAL(hipStreamCreate(&stream), hipSuccess);

  // copy src and tgt to device
  int *cudaSrc, *cudaTgt;
  BOOST_REQUIRE_EQUAL(
      hipMallocAsync(&cudaSrc, src.size() * sizeof(int), stream), hipSuccess);
  BOOST_REQUIRE_EQUAL(
      hipMallocAsync(&cudaTgt, tgt.size() * sizeof(int), stream), hipSuccess);
  BOOST_REQUIRE_EQUAL(
      hipMemcpyAsync(cudaSrc, src.data(), src.size() * sizeof(int),
                      hipMemcpyHostToDevice, stream),
      hipSuccess);
  BOOST_REQUIRE_EQUAL(
      hipMemcpyAsync(cudaTgt, tgt.data(), src.size() * sizeof(int),
                      hipMemcpyHostToDevice, stream),
      hipSuccess);

  // init label array
  int *cudaLabels;
  BOOST_REQUIRE_EQUAL(
      hipMallocAsync(&cudaLabels, nNodes * sizeof(int), stream), hipSuccess);

  // run connected components
  int cudaNumLabels = connectedComponentsCuda(src.size(), cudaSrc, cudaTgt,
                                              nNodes, cudaLabels, stream);
  BOOST_REQUIRE_EQUAL(hipStreamSynchronize(stream), hipSuccess);

  // print message from last cuda error code
  std::cout << "CUDA Error msg: " << hipGetErrorString(hipPeekAtLastError())
            << std::endl;
  BOOST_REQUIRE_EQUAL(hipGetLastError(), hipSuccess);

  // copy labels back
  std::vector<int> labelsFromCuda(nNodes);
  BOOST_REQUIRE_EQUAL(
      hipMemcpyAsync(labelsFromCuda.data(), cudaLabels, nNodes * sizeof(int),
                      hipMemcpyDeviceToHost, stream),
      hipSuccess);

  BOOST_REQUIRE_EQUAL(hipFreeAsync(cudaSrc, stream), hipSuccess);
  BOOST_REQUIRE_EQUAL(hipFreeAsync(cudaTgt, stream), hipSuccess);
  BOOST_REQUIRE_EQUAL(hipFreeAsync(cudaLabels, stream), hipSuccess);

  // sync
  BOOST_REQUIRE_EQUAL(hipStreamSynchronize(stream), hipSuccess);
  BOOST_REQUIRE_EQUAL(hipStreamDestroy(stream), hipSuccess);

  // print labelsFromCuda
  /*
      std::cout << "CUDA labels: ";
      for (int i = 0; i < nNodes; ++i) {
        std::cout << labelsFromCuda[i] << " ";
      }
      std::cout << std::endl;
  */
  // run boost graph for comparison

  BoostGraph G(nNodes);

  for (int i = 0; i < src.size(); ++i) {
    boost::add_edge(src[i], tgt[i], G);
  }

  std::vector<std::size_t> cpuLabels(boost::num_vertices(G));
  int cpuNumLabels = boost::connected_components(G, &cpuLabels[0]);

  // check
  BOOST_CHECK_EQUAL(cudaNumLabels, cpuNumLabels);
  BOOST_CHECK_EQUAL_COLLECTIONS(labelsFromCuda.begin(), labelsFromCuda.end(),
                                cpuLabels.begin(), cpuLabels.end());
}

BOOST_AUTO_TEST_CASE(full_test_tiny_graph) {
  auto [src, tgt] = makeRandomGraph(5, 10);
  testFullConnectedComponents(src, tgt);
}

BOOST_AUTO_TEST_CASE(full_test_small_graph) {
  auto [src, tgt] = makeRandomGraph(100, 500);
  testFullConnectedComponents(src, tgt);
}

BOOST_AUTO_TEST_CASE(full_test_big_graph) {
  for (int i = 0; i < 3; ++i) {
    std::cout << "Test graph " << i << std::endl;
    auto [src, tgt] = makeRandomGraph(100'000, 500'000);
    testFullConnectedComponents(src, tgt);
  }
}

BOOST_AUTO_TEST_CASE(test_from_file) {
  if (!std::filesystem::exists("edges_cuda_trackbuilding.txt")) {
    std::cout << "File edges_cuda_trackbuilding.txt not found" << std::endl;
    return;
  }

  std::ifstream file("edges_cuda_trackbuilding.txt");
  std::vector<int> src, tgt;
  int a, b;
  while (file >> a >> b) {
    src.push_back(a);
    tgt.push_back(b);
  }

  testFullConnectedComponents(src, tgt);
}

// try this pathologic case
BOOST_AUTO_TEST_CASE(special_1) {
  testFullConnectedComponents({1, 2}, {4, 7});
}

BOOST_AUTO_TEST_CASE(special_2) {
  Vi src{1, 2};
  Vi tgt{4, 7};
  checkLabeling(src, tgt);
}

BOOST_AUTO_TEST_CASE(special_3) {
  // clang-format off
  Vi labels      {0, 1, 2, 3, 1, 5, 6, 2};
  Vi refLabelMask{1, 1, 1, 1, 0, 1, 1, 0};
  Vi refPrefixSum{0, 1, 2, 3, 4, 4, 5, 6};
  Vi refLabels   {0, 1, 2, 3, 1, 4, 5, 2};
  // clang-format on

  testRelabeling(labels, refLabelMask, refPrefixSum, refLabels);
}

BOOST_AUTO_TEST_CASE(special_4) {
  Vi src{1, 2};
  Vi tgt{4, 7};

  auto labelsFromCuda = checkLabeling(src, tgt);

  Vi refLabelMask{1, 1, 1, 1, 0, 1, 1, 0};
  Vi refPrefixSum{0, 1, 2, 3, 4, 4, 5, 6};
  Vi refLabels{0, 1, 2, 3, 1, 4, 5, 2};

  testRelabeling(labelsFromCuda, refLabelMask, refPrefixSum, refLabels);
}
